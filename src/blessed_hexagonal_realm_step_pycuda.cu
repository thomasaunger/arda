#include "hip/hip_runtime.h"
__constant__ int DIMS = 3;

__constant__ int SYMMETRY_ORDER = 6;

__constant__ int NORTH     = 0;
__constant__ int NORTHEAST = 1;
__constant__ int SOUTHEAST = 2;
__constant__ int SOUTH     = 3;
__constant__ int SOUTHWEST = 4;
__constant__ int NORTHWEST = 5;

__constant__ int LEFT  = 1;
__constant__ int RIGHT = 2;

__constant__ int FORWARD = 1;

extern "C" {
  // Device helper function to rotate coordinates
  __device__ void RotateCoordinates(
    int kRadius,
    int orientation,
    int * loc_z,
    int * loc_y,
    int * loc_x
  ) {
    int loc_z_tmp = *loc_z - kRadius;
    int loc_y_tmp = *loc_y - kRadius;
    int loc_x_tmp = *loc_x - kRadius;
    if        (orientation == NORTH    ) {
      *loc_z = kRadius + loc_z_tmp;
      *loc_y = kRadius + loc_y_tmp;
      *loc_x = kRadius + loc_x_tmp;
    } else if (orientation == NORTHEAST) {
      *loc_z = kRadius - loc_y_tmp;
      *loc_y = kRadius - loc_x_tmp;
      *loc_x = kRadius - loc_z_tmp;
    } else if (orientation == SOUTHEAST) {
      *loc_z = kRadius + loc_x_tmp;
      *loc_y = kRadius + loc_z_tmp;
      *loc_x = kRadius + loc_y_tmp;
    } else if (orientation == SOUTH    ) {
      *loc_z = kRadius - loc_z_tmp;
      *loc_y = kRadius - loc_y_tmp;
      *loc_x = kRadius - loc_x_tmp;
    } else if (orientation == SOUTHWEST) {
      *loc_z = kRadius + loc_y_tmp;
      *loc_y = kRadius + loc_x_tmp;
      *loc_x = kRadius + loc_z_tmp;
    } else if (orientation == NORTHWEST) {
      *loc_z = kRadius - loc_x_tmp;
      *loc_y = kRadius - loc_z_tmp;
      *loc_x = kRadius - loc_y_tmp;
    }
  }

  // Device helper function to check whether a point is within bounds
  __device__ bool PointIsWithinBounds(
    int kRadius,
    int kSpaceLength,
    int loc_z,
    int loc_y,
    int loc_x
  ) {
    return (0 <= loc_z && loc_z < kSpaceLength &&
            0 <= loc_y && loc_y < kSpaceLength &&
            0 <= loc_x && loc_x < kSpaceLength &&
            loc_z + loc_y + loc_x == DIMS*kRadius);
  }

  // Device helper function to check whether a point is occupied
  __device__ bool PointIsOccupied(
    int * loc_z_arr,
    int * loc_y_arr,
    int * loc_x_arr,
    int kNumAgents,
    int kEnvId,
    int kThisAgentId,
    int loc_z,
    int loc_y,
    int loc_x
  ) {
    for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
      if (kAgentId != kThisAgentId) {
        int kAgentIdx = kEnvId * kNumAgents + kAgentId;
        if (loc_z_arr[kAgentIdx] == loc_z && loc_y_arr[kAgentIdx] == loc_y && loc_x_arr[kAgentIdx] == loc_x) {
          return true;
        }
      }
    }
    return false;
  }

  // Device helper function to generate an unoccupied point
  __device__ void GenerateUnoccupiedPoint(
    int kRadius,
    int kSpaceLength,
    int * loc_z_arr,
    int * loc_y_arr,
    int * loc_x_arr,
    int kNumAgents,
    int kEnvId,
    int * loc_z,
    int * loc_y,
    int * loc_x
  ) {
    // Use last agent's state to generate a random point
    hiprandState_t* state = states[kEnvId * kNumAgents + kNumAgents - 1];
    do {
      // Generate random coordinates from uniform distribution over [0, kSpaceLength - 1]
      *loc_y = kSpaceLength * (1.0 - hiprand_uniform(state));
      *loc_x = kSpaceLength * (1.0 - hiprand_uniform(state));
      *loc_z = DIMS*kRadius - *loc_y - *loc_x;
    } while (!PointIsWithinBounds(
      kRadius,
      kSpaceLength,
      *loc_z,
      *loc_y,
      *loc_x
    ) || PointIsOccupied(
      loc_z_arr,
      loc_y_arr,
      loc_x_arr,
      kNumAgents,
      kEnvId,
      -1,
      *loc_z,
      *loc_y,
      *loc_x
    ));
  }

  // Device helper function to compute rewards
  __device__ void CudaBlessedRealmComputeReward(
    int * loc_z_arr,
    int * loc_y_arr,
    int * loc_x_arr,
    int * goal_point_arr,
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // Initialize rewards
      rewards_arr[kThisAgentArrayIdx] = 0.0;

      // Check whether the agent has reached the goal
      if (loc_z_arr[kThisAgentArrayIdx] == goal_point_arr[kEnvId * DIMS    ] &&
          loc_y_arr[kThisAgentArrayIdx] == goal_point_arr[kEnvId * DIMS + 1] &&
          loc_x_arr[kThisAgentArrayIdx] == goal_point_arr[kEnvId * DIMS + 2]) {
        rewards_arr[kThisAgentArrayIdx] = 1.0 * (1.0 - env_timestep_arr[kEnvId] / float(kEpisodeLength));
        // done_arr[kEnvId] = 1;
      }

      // Use only last agent's thread to check whether the maximum number of timesteps has been reached
      if (kThisAgentId == kNumAgents - 1) {
        if (env_timestep_arr[kEnvId] == kEpisodeLength) {
            done_arr[kEnvId] = 1;
        }
      }
    }
  }

  // Device helper function to generate observation
  __device__ void CudaBlessedRealmGenerateObservation(
    int kRadius,
    int kSpaceLength,
    int * loc_z_arr,
    int * loc_y_arr,
    int * loc_x_arr,
    int * orientation_arr,
    int * goal_point_arr,
    float * obs_arr,
    int * done_arr,
    int * env_timestep_arr,
    const int kNumAgents,
    const int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // obs shape is (num_envs, kNumAgents, n)
      const int n = 7;
      const int kThisAgentIdxOffset = (kEnvId * kNumAgents + kThisAgentId) * n;

      // Initialize obs
      for (int i = 0; i < n; i++) {
        obs_arr[kThisAgentIdxOffset + i] = 0.0;
      }

      int kAgentIdx;
      int loc_z;
      int loc_y;
      int loc_x;
      if (done_arr[kEnvId]) {
        // Reinitialize agent points
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          loc_z_arr[kAgentIdx] = -1;
          loc_y_arr[kAgentIdx] = -1;
          loc_x_arr[kAgentIdx] = -1;
        }

        // Reset agent points
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          GenerateUnoccupiedPoint(
            kRadius,
            kSpaceLength,
            loc_z_arr,
            loc_y_arr,
            loc_x_arr,
            kNumAgents,
            kEnvId,
            &loc_z,
            &loc_y,
            &loc_x
          );
          loc_z_arr[kAgentIdx] = loc_z;
          loc_y_arr[kAgentIdx] = loc_y;
          loc_x_arr[kAgentIdx] = loc_x;
        }

        // Reset goal point
        GenerateUnoccupiedPoint(
          kRadius,
          kSpaceLength,
          loc_z_arr,
          loc_y_arr,
          loc_x_arr,
          kNumAgents,
          kEnvId,
          &loc_z,
          &loc_y,
          &loc_x
        );
        goal_point_arr[kEnvId * DIMS    ] = loc_z;
        goal_point_arr[kEnvId * DIMS + 1] = loc_y;
        goal_point_arr[kEnvId * DIMS + 2] = loc_x;

        // Reset agent orientations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          // Use agent's state to generate a random orientation
          hiprandState_t* state = states[kAgentIdx];
          // Generate a random orientation from uniform distribution over [0, SYMMETRY_ORDER - 1]
          orientation_arr[kAgentIdx] = SYMMETRY_ORDER * (1.0 - hiprand_uniform(state));
        }
      }

      loc_z = loc_z_arr[kThisAgentArrayIdx];
      loc_y = loc_y_arr[kThisAgentArrayIdx];
      loc_x = loc_x_arr[kThisAgentArrayIdx];
      RotateCoordinates(kRadius, orientation_arr[kThisAgentArrayIdx], &loc_z, &loc_y, &loc_x);
      obs_arr[kThisAgentIdxOffset    ] = loc_z;
      obs_arr[kThisAgentIdxOffset + 1] = loc_y;
      obs_arr[kThisAgentIdxOffset + 2] = loc_x;

      loc_z = goal_point_arr[kEnvId * DIMS    ];
      loc_y = goal_point_arr[kEnvId * DIMS + 1];
      loc_x = goal_point_arr[kEnvId * DIMS + 2];
      RotateCoordinates(kRadius, orientation_arr[kThisAgentArrayIdx], &loc_z, &loc_y, &loc_x);
      obs_arr[kThisAgentIdxOffset + 3] = loc_z - obs_arr[kThisAgentIdxOffset    ];
      obs_arr[kThisAgentIdxOffset + 4] = loc_y - obs_arr[kThisAgentIdxOffset + 1];
      obs_arr[kThisAgentIdxOffset + 5] = loc_x - obs_arr[kThisAgentIdxOffset + 2];
      obs_arr[kThisAgentIdxOffset + 6] = orientation_arr[kThisAgentArrayIdx];
    }
  }

  __global__ void CudaBlessedRealmStep(
    const bool kMarred,
    int kRadius,
    int * loc_z_arr,
    int * loc_y_arr,
    int * loc_x_arr,
    int * orientation_arr,
    int * agent_types_arr,
    int * goal_point_arr,
    float * obs_arr,
    int * action_indices_arr,
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength
  ) {
    const int kEnvId = getEnvID(blockIdx.x);
    const int kThisAgentId = getAgentID(threadIdx.x, blockIdx.x, blockDim.x);
    const int kThisAgentArrayIdx = kEnvId * kNumAgents + kThisAgentId;
    const int kNumActions = 2;
    const int kThisAgentActionIdxOffset = kEnvId * kNumAgents * kNumActions + kThisAgentId * kNumActions;
    const int kSpaceLength = 2*kRadius + 1;

    int action_turn = action_indices_arr[kThisAgentActionIdxOffset    ];
    int action_move = action_indices_arr[kThisAgentActionIdxOffset + 1];

    int loc_z_tmp = loc_z_arr[kThisAgentArrayIdx];
    int loc_y_tmp = loc_y_arr[kThisAgentArrayIdx];
    int loc_x_tmp = loc_x_arr[kThisAgentArrayIdx];

    if (action_move == FORWARD) {
      if        (orientation_arr[kThisAgentArrayIdx] == NORTH    ) {
        loc_z_tmp += 1;
        loc_y_tmp -= 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == NORTHEAST) {
        loc_y_tmp -= 1;
        loc_x_tmp += 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == SOUTHEAST) {
        loc_z_tmp -= 1;
        loc_x_tmp += 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == SOUTH    ) {
        loc_z_tmp -= 1;
        loc_y_tmp += 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == SOUTHWEST) {
        loc_y_tmp += 1;
        loc_x_tmp -= 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == NORTHWEST) {
        loc_z_tmp += 1;
        loc_x_tmp -= 1;
      }
    } else if (action_turn == LEFT) {
      orientation_arr[kThisAgentArrayIdx] = (orientation_arr[kThisAgentArrayIdx] + SYMMETRY_ORDER - 1) % SYMMETRY_ORDER;
    } else if (action_turn == RIGHT) {
      orientation_arr[kThisAgentArrayIdx] = (orientation_arr[kThisAgentArrayIdx] +                  1) % SYMMETRY_ORDER;
    }

    if (
      PointIsWithinBounds(
        kRadius,
        kSpaceLength,
        loc_z_tmp,
        loc_y_tmp,
        loc_x_tmp
      )  // && !PointIsOccupied(
      //   loc_z_arr,
      //   loc_y_arr,
      //   loc_x_arr,
      //   kNumAgents,
      //   kEnvId,
      //   kThisAgentId,
      //   loc_z_tmp,
      //   loc_y_tmp,
      //   loc_x_tmp
      // )
    ) {
      // Update the point of the agent
      loc_z_arr[kThisAgentArrayIdx] = loc_z_tmp;
      loc_y_arr[kThisAgentArrayIdx] = loc_y_tmp;
      loc_x_arr[kThisAgentArrayIdx] = loc_x_tmp;
    }

    // Wait here until timestep has been updated
    if (kThisAgentId == 0) {
      env_timestep_arr[kEnvId] += 1;
    }

    // Make sure all agents have updated their states
    __sync_env_threads();

    assert(0 < env_timestep_arr[kEnvId] && env_timestep_arr[kEnvId] <= kEpisodeLength);

    // -------------------------------
    // Compute reward
    // -------------------------------
    CudaBlessedRealmComputeReward(
      loc_z_arr,
      loc_y_arr,
      loc_x_arr,
      goal_point_arr,
      rewards_arr,
      done_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx
    );

    // -------------------------------
    // Generate observation
    // -------------------------------
    CudaBlessedRealmGenerateObservation(
      kRadius,
      kSpaceLength,
      loc_z_arr,
      loc_y_arr,
      loc_x_arr,
      orientation_arr,
      goal_point_arr,
      obs_arr,
      done_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx
    );
  }
}
