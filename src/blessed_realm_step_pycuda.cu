#include "hip/hip_runtime.h"
__constant__ int NUM_COORDINATES = 2;

__constant__ int NUM_ORIENTATIONS = 4;

__constant__ int NORTH = 0;
__constant__ int EAST  = 1;
__constant__ int SOUTH = 2;
__constant__ int WEST  = 3;

__constant__ int LEFT  = 1;
__constant__ int RIGHT = 2;

__constant__ int FORWARD = 1;

extern "C" {
  // Device helper function to rotate coordinates
  __device__ void RotateCoordinates(
    int kGridLength,
    int * loc_y,
    int * loc_x,
    int orientation
  ) {
    int loc_y_tmp = *loc_y;
    int loc_x_tmp = *loc_x;
    if (orientation == NORTH) {
      *loc_y = loc_y_tmp;
      *loc_x = loc_x_tmp;
    } else if (orientation == EAST) {
      *loc_y = kGridLength - 1 - loc_x_tmp;
      *loc_x = loc_y_tmp;
    } else if (orientation == SOUTH) {
      *loc_y = kGridLength - 1 - loc_y_tmp;
      *loc_x = kGridLength - 1 - loc_x_tmp;
    } else if (orientation == WEST) {
      *loc_y = loc_x_tmp;
      *loc_x = kGridLength - 1 - loc_y_tmp;
    }
  }

  // Device helper function to check whether a location is within bounds
  __device__ bool LocationIsWithinBounds(
    int kGridLength,
    int loc_y,
    int loc_x
  ) {
    return (0 <= loc_y && loc_y < kGridLength && 0 <= loc_x && loc_x < kGridLength);
  }

  // Device helper function to check whether a location is occupied
  __device__ bool LocationIsOccupied(
    int * loc_y_arr,
    int * loc_x_arr,
    int kNumAgents,
    int kEnvId,
    int kThisAgentId,
    int loc_y,
    int loc_x
  ) {
    for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
      if (kAgentId != kThisAgentId) {
        int kAgentIdx = kEnvId * kNumAgents + kAgentId;
        if (loc_y_arr[kAgentIdx] == loc_y && loc_x_arr[kAgentIdx] == loc_x) {
          return true;
        }
      }
    }
    return false;
  }

  // Device helper function to generate an unoccupied location
  __device__ void GenerateUnoccupiedLocation(
    int kGridLength,
    int * loc_y_arr,
    int * loc_x_arr,
    int kNumAgents,
    int kEnvId,
    int * loc_y,
    int * loc_x
  ) {
    // Use last agent's state to generate a random location
    hiprandState_t* state = states[kEnvId * kNumAgents + kNumAgents - 1];
    do {
      // Generate random coordinates from uniform distribution over [0, kGridLength - 1]
      *loc_y = kGridLength * (1.0 - hiprand_uniform(state));
      *loc_x = kGridLength * (1.0 - hiprand_uniform(state));
    } while (LocationIsOccupied(
      loc_y_arr,
      loc_x_arr,
      kNumAgents,
      kEnvId,
      -1,
      *loc_y,
      *loc_x
    ));
  }

  // Device helper function to compute rewards
  __device__ void CudaBlessedRealmComputeReward(
    int * loc_y_arr,
    int * loc_x_arr,
    int * goal_location_arr,
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // Initialize rewards
      rewards_arr[kThisAgentArrayIdx] = 0.0;

      // Check whether the agent has reached the goal
      if (loc_y_arr[kThisAgentArrayIdx] == goal_location_arr[kEnvId * NUM_COORDINATES    ] &&
          loc_x_arr[kThisAgentArrayIdx] == goal_location_arr[kEnvId * NUM_COORDINATES + 1]) {
        rewards_arr[kThisAgentArrayIdx] = 1.0 * (1.0 - env_timestep_arr[kEnvId] / float(kEpisodeLength));
        // done_arr[kEnvId] = 1;
      }

      // Use only last agent's thread to check whether the maximum number of timesteps has been reached
      if (kThisAgentId == kNumAgents - 1) {
        if (env_timestep_arr[kEnvId] == kEpisodeLength - 1) {
            done_arr[kEnvId] = 1;
        }
      }
    }
  }

  // Device helper function to generate observation
  __device__ void CudaBlessedRealmGenerateObservation(
    int kGridLength,
    int * loc_y_arr,
    int * loc_x_arr,
    int * orientation_arr,
    int * goal_location_arr,
    float * obs_arr,
    int * done_arr,
    int * env_timestep_arr,
    const int kNumAgents,
    const int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // obs shape is (num_envs, kNumAgents, 1 + kGridLength * kGridLength + kNumAgents)
      const int kThisAgentIdxOffset = (kEnvId * kNumAgents + kThisAgentId) * (1 + kGridLength * kGridLength + kNumAgents);

      // Initialize obs
      for (int kLocationIdx = 0; kLocationIdx < kGridLength * kGridLength; kLocationIdx++) {
        obs_arr[kThisAgentIdxOffset + 1 + kLocationIdx] = 0.0;
      }

      int kAgentIdx;
      int loc_y;
      int loc_x;
      if (done_arr[kEnvId]) {
        // Reinitialize agent locations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          loc_y_arr[kAgentIdx] = -1;
          loc_x_arr[kAgentIdx] = -1;
        }

        // Reset agent locations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          GenerateUnoccupiedLocation(
            kGridLength,
            loc_y_arr,
            loc_x_arr,
            kNumAgents,
            kEnvId,
            &loc_y,
            &loc_x
          );
          loc_y_arr[kAgentIdx] = loc_y;
          loc_x_arr[kAgentIdx] = loc_x;
        }

        // Reset goal location
        GenerateUnoccupiedLocation(
          kGridLength,
          loc_y_arr,
          loc_x_arr,
          kNumAgents,
          kEnvId,
          &loc_y,
          &loc_x
        );
        goal_location_arr[kEnvId * NUM_COORDINATES    ] = loc_y;
        goal_location_arr[kEnvId * NUM_COORDINATES + 1] = loc_x;

        // Reset agent orientations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          // Use agent's state to generate a random orientation
          hiprandState_t* state = states[kAgentIdx];
          // Generate a random orientation from uniform distribution over [0, NUM_ORIENTATIONS - 1]
          orientation_arr[kAgentIdx] = NUM_ORIENTATIONS * (1.0 - hiprand_uniform(state));
        }
      }

      loc_y = goal_location_arr[kEnvId * NUM_COORDINATES    ];
      loc_x = goal_location_arr[kEnvId * NUM_COORDINATES + 1];
      RotateCoordinates(kGridLength, &loc_y, &loc_x, orientation_arr[kThisAgentArrayIdx]);
      obs_arr[kThisAgentIdxOffset + 1 + loc_y * kGridLength + loc_x] = 1;

      for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
        kAgentIdx = kEnvId * kNumAgents + kAgentId;

        // Add agent location
        loc_y = loc_y_arr[kAgentIdx];
        loc_x = loc_x_arr[kAgentIdx];
        RotateCoordinates(kGridLength, &loc_y, &loc_x, orientation_arr[kThisAgentArrayIdx]);
        obs_arr[kThisAgentIdxOffset + 1 + loc_y * kGridLength + loc_x] = kAgentIdx + 2;

        // Set agent orientation
        obs_arr[kThisAgentIdxOffset + 1 + kGridLength * kGridLength + kAgentId] = (orientation_arr[kAgentIdx] + NUM_ORIENTATIONS - orientation_arr[kThisAgentArrayIdx]) % NUM_ORIENTATIONS;
      }
    }
  }

  // Device helper function to generate observation
  __device__ void CudaBlessedRealmGenerateObservation2(
    int kGridLength,
    int * loc_y_arr,
    int * loc_x_arr,
    int * orientation_arr,
    int * goal_location_arr,
    float * obs_arr,
    int * done_arr,
    int * env_timestep_arr,
    const int kNumAgents,
    const int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // obs shape is (num_envs, kNumAgents, 5)
      const int kThisAgentIdxOffset = (kEnvId * kNumAgents + kThisAgentId) * 5;

      // Initialize obs
      for (int i = 0; i < 5; i++) {
        obs_arr[kThisAgentIdxOffset + i] = 0.0;
      }

      int kAgentIdx;
      int loc_y;
      int loc_x;
      if (done_arr[kEnvId]) {
        // Reinitialize agent locations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          loc_y_arr[kAgentIdx] = -1;
          loc_x_arr[kAgentIdx] = -1;
        }

        // Reset agent locations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          GenerateUnoccupiedLocation(
            kGridLength,
            loc_y_arr,
            loc_x_arr,
            kNumAgents,
            kEnvId,
            &loc_y,
            &loc_x
          );
          loc_y_arr[kAgentIdx] = loc_y;
          loc_x_arr[kAgentIdx] = loc_x;
        }

        // Reset goal location
        GenerateUnoccupiedLocation(
          kGridLength,
          loc_y_arr,
          loc_x_arr,
          kNumAgents,
          kEnvId,
          &loc_y,
          &loc_x
        );
        goal_location_arr[kEnvId * NUM_COORDINATES    ] = loc_y;
        goal_location_arr[kEnvId * NUM_COORDINATES + 1] = loc_x;

        // Reset agent orientations
        for (int kAgentId = 0; kAgentId < kNumAgents; kAgentId++) {
          kAgentIdx = kEnvId * kNumAgents + kAgentId;
          // Use agent's state to generate a random orientation
          hiprandState_t* state = states[kAgentIdx];
          // Generate a random orientation from uniform distribution over [0, NUM_ORIENTATIONS - 1]
          orientation_arr[kAgentIdx] = NUM_ORIENTATIONS * (1.0 - hiprand_uniform(state));
        }
      }

      obs_arr[kThisAgentIdxOffset    ] = loc_y_arr[kThisAgentArrayIdx];
      obs_arr[kThisAgentIdxOffset + 1] = loc_x_arr[kThisAgentArrayIdx];
      obs_arr[kThisAgentIdxOffset + 2] = orientation_arr[kThisAgentArrayIdx];
      obs_arr[kThisAgentIdxOffset + 3] = goal_location_arr[kEnvId * NUM_COORDINATES    ];
      obs_arr[kThisAgentIdxOffset + 4] = goal_location_arr[kEnvId * NUM_COORDINATES + 1];
    }
  }

  __global__ void CudaBlessedRealmStep(
    const bool kMarred,
    int kGridLength,
    int * loc_y_arr,
    int * loc_x_arr,
    int * orientation_arr,
    int * agent_types_arr,
    int * goal_location_arr,
    float * obs_arr,
    int * action_indices_arr,
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength
  ) {
    const int kEnvId = getEnvID(blockIdx.x);
    const int kThisAgentId = getAgentID(threadIdx.x, blockIdx.x, blockDim.x);
    const int kThisAgentArrayIdx = kEnvId * kNumAgents + kThisAgentId;
    const int kNumActions = 2;
    const int kThisAgentActionIdxOffset = kEnvId * kNumAgents * kNumActions + kThisAgentId * kNumActions;

    int action_turn = action_indices_arr[kThisAgentActionIdxOffset    ];
    int action_move = action_indices_arr[kThisAgentActionIdxOffset + 1];

    int loc_y_tmp = loc_y_arr[kThisAgentArrayIdx];
    int loc_x_tmp = loc_x_arr[kThisAgentArrayIdx];

    if (action_move == FORWARD) {
      if        (orientation_arr[kThisAgentArrayIdx] == NORTH) {
        loc_y_tmp -= 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == EAST ) {
        loc_x_tmp += 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == SOUTH) {
        loc_y_tmp += 1;
      } else if (orientation_arr[kThisAgentArrayIdx] == WEST ) {
        loc_x_tmp -= 1;
      }
    } else if (action_turn == LEFT) {
      orientation_arr[kThisAgentArrayIdx] = (orientation_arr[kThisAgentArrayIdx] + NUM_ORIENTATIONS - 1) % NUM_ORIENTATIONS;
    } else if (action_turn == RIGHT) {
      orientation_arr[kThisAgentArrayIdx] = (orientation_arr[kThisAgentArrayIdx] +                    1) % NUM_ORIENTATIONS;
    }

    if (
      LocationIsWithinBounds(
        kGridLength,
        loc_y_tmp,
        loc_x_tmp
      ) && !LocationIsOccupied(
        loc_y_arr,
        loc_x_arr,
        kNumAgents,
        kEnvId,
        kThisAgentId,
        loc_y_tmp,
        loc_x_tmp
      )
    ) {
      // Update the location of the agent
      loc_y_arr[kThisAgentArrayIdx] = loc_y_tmp;
      loc_x_arr[kThisAgentArrayIdx] = loc_x_tmp;
    }

    // assert(env_timestep_arr[kEnvId] > 0 && env_timestep_arr[kEnvId] <= kEpisodeLength);

    // Make sure all agents have updated their states
    __sync_env_threads();

    // -------------------------------
    // Compute reward
    // -------------------------------
    CudaBlessedRealmComputeReward(
      loc_y_arr,
      loc_x_arr,
      goal_location_arr,
      rewards_arr,
      done_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx
    );

    // -------------------------------
    // Generate observation
    // -------------------------------
    CudaBlessedRealmGenerateObservation2(
      kGridLength,
      loc_y_arr,
      loc_x_arr,
      orientation_arr,
      goal_location_arr,
      obs_arr,
      done_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx
    );

    // Increment time ONCE -- only 1 thread can do this.
    if (kThisAgentId == kNumAgents - 1) {
      env_timestep_arr[kEnvId] += 1;
    }
  }
}
