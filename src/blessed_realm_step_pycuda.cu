#include "hip/hip_runtime.h"
__constant__ float kPi = 3.141592654;
__constant__ float kTwoPi = 6.283185308;
__constant__ float kEpsilon = 1.0e-10;  // to prevent indeterminate cases

extern "C" {
  // Device helper function to generate observation
  __device__ void CudaBlessedRealmGenerateObservation(
    float * obs_arr,
    int * env_timestep_arr,
    const int kNumAgents,
    const int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    int num_features = 7;

    if (kThisAgentId < kNumAgents) {
      // obs shape is (num_envs, kNumAgents,
      // num_features * (kNumAgents - 1) + 1)
      const int kThisAgentIdxOffset = kEnvId * kNumAgents *
        (num_features * (kNumAgents - 1) + 1) +
        kThisAgentId * (num_features * (kNumAgents - 1) + 1);
      // Initialize obs
      int index = 0;
      for (int other_agent_id = 0; other_agent_id < kNumAgents;
      other_agent_id++) {
        if (other_agent_id != kThisAgentId) {
          obs_arr[kThisAgentIdxOffset + 0 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 1 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 2 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 3 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 4 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 5 * (kNumAgents - 1) + index]
            = 0.0;
          obs_arr[kThisAgentIdxOffset + 6 * (kNumAgents - 1) + index]
            = 0.0;
          index += 1;
        }
      }
      obs_arr[kThisAgentIdxOffset + num_features * (kNumAgents - 1)] = 0.0;
    }
  }

  // Device helper function to compute rewards
  __device__ void CudaBlessedRealmComputeReward(
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength,
    const int kEnvId,
    const int kThisAgentId,
    const int kThisAgentArrayIdx
  ) {
    if (kThisAgentId < kNumAgents) {
      // initialize rewards
      rewards_arr[kThisAgentArrayIdx] = 0.0; 

      // Wait here to update the number of runners before determining done_arr
      __sync_env_threads();
      // Use only agent 0's thread to set done_arr
      if (kThisAgentId == 0) {
        if (env_timestep_arr[kEnvId] == kEpisodeLength) {
            done_arr[kEnvId] = 1;
        }
      }
    }
  }

  __global__ void CudaBlessedRealmStep(
    const bool kMarred,
    int kGridLength,
    int * loc_x_arr,
    int * loc_y_arr,
    int * orientation_arr,
    float * obs_arr,
    int * action_indices_arr,
    float * rewards_arr,
    int * done_arr,
    int * env_timestep_arr,
    int kNumAgents,
    int kEpisodeLength
  ) {
    const int kEnvId = getEnvID(blockIdx.x);
    const int kThisAgentId = getAgentID(threadIdx.x, blockIdx.x, blockDim.x);
    const int kThisAgentArrayIdx = kEnvId * kNumAgents + kThisAgentId;
    const int kNumActions = 2;

    // Increment time ONCE -- only 1 thread can do this.
    if (kThisAgentId == 0) {
      env_timestep_arr[kEnvId] += 1;
    }

    // Generate a random float between 0 and 1.
    hiprandState_t* state = states[kThisAgentArrayIdx];  // Retrieve the state for this thread/agent.
    float randomValue = hiprand_uniform(state);  // Generate the random float.

    // Wait here until timestep has been updated
    __sync_env_threads();

    assert(env_timestep_arr[kEnvId] > 0 && env_timestep_arr[kEnvId] <=
      kEpisodeLength);

    // Make sure all agents have updated their states
    __sync_env_threads();
    // -------------------------------
    // Generate observation
    // -------------------------------
    CudaBlessedRealmGenerateObservation(
      obs_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx);

    // -------------------------------
    // Compute reward
    // -------------------------------
    CudaBlessedRealmComputeReward(
      rewards_arr,
      done_arr,
      env_timestep_arr,
      kNumAgents,
      kEpisodeLength,
      kEnvId,
      kThisAgentId,
      kThisAgentArrayIdx);
  }
}
